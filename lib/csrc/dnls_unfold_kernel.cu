#include "hip/hip_runtime.h"



// #include <torch/extension.h>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

/****************************

       Helper Funcs

****************************/

#define CUDA_KERNEL_LOOP_TYPE(i, n, index_type)                         \
  int64_t _i_n_d_e_x = blockIdx.x * blockDim.x + threadIdx.x;           \
  for (index_type i=_i_n_d_e_x; _i_n_d_e_x < (n); _i_n_d_e_x+=blockDim.x * gridDim.x, i=_i_n_d_e_x)

#define CUDA_KERNEL_LOOP(i, n) CUDA_KERNEL_LOOP_TYPE(i, n, int)

__inline__ __device__ int bounds(int val, int lim ){
  if (val < 0){
    val = -val;
  }else if (val >= lim){
    val = 2*lim - val - 2;
  }
  return val;
}

/****************************

       Forward Pass

****************************/

template <typename scalar_t>
__global__ void dnls_unfold_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> patches,
    int qStart, int qStride, int dilation, int qpt, int kpt) {

    // -- shapes --
    int nframes = vid.size(0);
    int colors = vid.size(1);
    int height = vid.size(2);
    int width = vid.size(3);
    int nq = patches.size(0);
    int k = patches.size(1);
    int pt = patches.size(2);
    int ps = patches.size(4);
    int psHalf = (int)ps/2;
    int heigh_width = height*width;

    // -- cuda threads --
    int pi = threadIdx.y;
    int pj = threadIdx.z;

    // -- batching --
    int query_start_block = blockIdx.x*qpt;
    int k_start = threadIdx.x*kpt;

    // inits
    int qIndex;
    int qi,ki,ti,hi,wi;
    int vi_h,vi_w,vi_t;
    bool valid_hw,valid_t,valid;
    scalar_t pix;

    // -- range --
    for(int _qi = 0; _qi < qpt; _qi++){

      // -- query index --
      qi = _qi + query_start_block + qStart;
      if (qi >= nq){ continue; }

      for(int _ki = 0; _ki < kpt; _ki++){

        // -- k index --
        ki = k_start + _ki;
        if (ki >= k){ continue; }

        // -- fill --
        qIndex = qi*qStride;
        wi = qIndex % width;
        hi = (qIndex/width) % height;
        ti = (qIndex/heigh_width) % nframes;

        // -- fill across cuda threads --
        vi_h = bounds(hi+dilation*(pi - psHalf),height);
        vi_w = bounds(wi+dilation*(pj - psHalf),width);

        // -- spatially valid --
        valid_hw = (vi_h >= 0) && (vi_h < height);
        valid_hw = valid_hw && (vi_w >= 0) && (vi_w < width);

        // -- iterate over loop --
        for(int pk = 0; pk < pt; pk++){

          // -- check valid --
          vi_t = bounds(ti + pk,nframes);
          valid_t = (vi_t >= 0) && (vi_t < nframes);
          valid = valid_hw && valid_t;

          // -- colors --
          for(int ci = 0; ci < colors; ci++){
            if (valid){
              pix = vid[vi_t][ci][vi_h][vi_w];
            }else{
              pix = 0.;
            }
            patches[qi][ki][pk][ci][pi][pj] = pix;
          }
        }
      }
    }
}

void dnls_cuda_unfold_forward(
    torch::Tensor vid, torch::Tensor patches,
    int qStart, int qStride, int dilation){

  // -- kernel blocks --
  int numQueries = patches.size(0);
  int k = 1;
  int qpt = 10;
  int nblocks = (numQueries-1)/qpt+1;
  int pt = patches.size(2);
  assert(pt == 1);

  // -- kernel threads --
  int ps = patches.size(5);
  int MAX_THREADS = 1024;
  int dim = ps*ps;
  int kpb = MAX_THREADS/dim; // num of "k" managed per block
  int kpt = ((k - 1)/kpb) + 1; // num of "k" per thread
  dim3 nthreads(kpb,ps,ps);

  // launch kernel
  AT_DISPATCH_FLOATING_TYPES(patches.type(), "dnls_unfold_forward_kernel", ([&] {
    dnls_unfold_forward_kernel<scalar_t><<<nblocks, nthreads>>>(
        vid.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        qStart,qStride,dilation,qpt,kpt);
      }));
}


/****************************

       Backward Pass

****************************/

template <typename scalar_t>
__global__ void dnls_unfold_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> vid,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> patches,
    int iStart, int qStart, int stride, int dilation, int num_kernels) {

    // -- unpack --
    int nframes = vid.size(0);
    int colors = vid.size(1);
    int height = vid.size(2);
    int width = vid.size(3);
    int pt = patches.size(2);
    int ps = patches.size(5);
    int numQueries = patches.size(0);
    int psHalf = ps/2;
    int hw = height*width;
    bool valid,valid_q,is_edge;
    int nhits,nhits_q;
    // int ndim = ps*ps*pt;

    CUDA_KERNEL_LOOP(_index, num_kernels) {

      int index = (_index);// + iStart);
      const int64_t w_im = index % width;
      const int64_t h_im = (index / width) % height;
      const int64_t t_im = (index / hw);

      // -- allow partial nhits if edge --
      // int padf = dilation*ps;
      // bool is_edge = (w_im < padf) || (w_im > (width-padf));
      // is_edge = is_edge || (h_im < padf) || (h_im > (height-padf));
        
      for(int ci = 0; ci < colors; ci++){
        nhits = 0;
        nhits_q = 0;
        scalar_t val = 0;
        for (int pk = 0; pk < pt; pk++){
          for (int pi = 0; pi < ps; pi++){
            for (int pj = 0; pj < ps; pj++){

              // -- offsets for ni --
              int _wi = w_im + dilation*(pi - psHalf);
              int _hi = h_im + dilation*(pj - psHalf);
              int ti = t_im + pk;

              // -- check bounds --
              // NOTE; this will not work for dilation > 1
              valid = (_wi >= -psHalf) && (_wi < (width+psHalf));
              valid = valid && (_hi >= -psHalf) && (_hi < (height+psHalf));
              int wi = bounds(_wi,width);
              int hi = bounds(_hi,height);

              // -- compute ni --
              int qi = ti * hw + hi * width + wi; // maybe stride here?
              qi -= qStart;

              // -- patch indexing --
              int w_ip = ps-1-pi;
              int h_ip = ps-1-pj;

              // -- reflect to match --
              if (_wi > wi){
                w_ip = pi;
                valid = valid && (w_ip < psHalf);
              }
              else if(_wi < wi){
                w_ip = pi;
                valid = valid && (w_ip > psHalf);
              }

              if (_hi > hi){
                h_ip = pj;
                valid = valid && (h_ip < psHalf);
              }
              else if(_hi < hi){
                h_ip = pj;
                valid = valid && (h_ip > psHalf);
              }

              // -- accumulate --
              valid_q = valid && (qi >= 0) && (qi < numQueries);
              if (valid_q){
                val += patches[qi][0][0][ci][h_ip][w_ip];
                nhits_q += 1;
              }
              if(valid){
                nhits += 1;
              }

            }
          } // for patch size
        } // for patch size
        bool eq_hits = nhits == nhits_q;
        // bool hit_req = true;//((not is_edge) && (nhits == ndim)) || is_edge;
        if (eq_hits){
          vid[t_im][ci][h_im][w_im] =  val;
        }
      } // for colors
    }
}

void dnls_cuda_unfold_backward(
  torch::Tensor grad_vid,torch::Tensor patches,
  int qStart, int qStride, int dilation) {

  // -- kernel blocks --
  // int numQueries = patches.size(0);
  // int k = 1;
  // int nframes = grad_vid.size(0);
  // int height = grad_vid.size(0);
  // int width = grad_vid.size(0);
  int nthreads = 512;
  int num_kernels = patches.size(0);//nframes*height*width;
  int nblocks = (num_kernels-1) / nthreads+1;

  // get starting pixel
  int iStart = qStart; // some actual logic goes here; what is the "min" pixel (top-left)

  // -- kernel threads --
  // int ps = patches.size(5);
  // int MAX_THREADS = 1024;
  // int dim = ps*ps;
  // int kpb = MAX_THREADS/dim; // num of "k" managed per block
  // int kpt = ((k - 1)/kpb) + 1; // num of "k" per thread
  // dim3 nthreads(kpb,ps,ps);

  // -- launch kernel --
  AT_DISPATCH_FLOATING_TYPES(patches.type(), "dnls_unfold_backward_kernel", ([&] {
    dnls_unfold_backward_kernel<scalar_t>
      <<<nblocks, nthreads>>>(
        grad_vid.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>(),
        iStart,qStart,qStride,dilation,num_kernels);
  }));

}
